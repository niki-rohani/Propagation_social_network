
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipblas.h"

#define __CUDA__NBTHREADS 1024

extern "C"
__global__ void functionShrink(const float *A,float *B, int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) 
	{
		if ((A[i]>=-1) && (A[i]<=1)) B[i]=0; 
		else B[i]=1;
	}
}

__global__ void functionDShrink(const float *A,float *B, int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) 
	{
		if ((A[i]>=-1) && (A[i]<=1)) B[i]=0; 		
	}
}

__global__ void functionPShrink(const float *A,float *B, int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) 
	{
		if (A[i]<0) B[i]=0;
		else B[i]=A[i];
	}
}

__global__ void functionDPShrink(const float *A,float *B, int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) 
	{
		if (A[i]<0) B[i]=0;
	}
}


__global__ void functionAdaptedTanH(const float *A,float *B, int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) B[i]=(float)(1.7159*tanh(0.6666*A[i]));
}

__global__ void functionLogistic(const float *A,float *B, int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) B[i]=(float)(1.0/(1.0+exp(-A[i])));
}

__global__ void functionDAdaptedTanH(const float *A,float *B, int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	
	float t=(float)(tanh(0.6666*A[i]));
	if (i<N) B[i]=(float)(0.66666*1.7159*(1.0-t*t));
}

__global__ void functionDLogistic(const float *A,float *B, int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	
	float f=(float)(1.0/(1.0+exp(-A[i])));
	if (i<N) B[i]=(float)(f*(1.0-f));
}

__global__ void kerrorSquare(float *A,const float *z,const float *x,const float *y,int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) A[i]=z[i]*(x[i]-y[i]);
}


__global__ void functionHingeLoss(const float *A,float *B, int N)
{	
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	float f=A[i]*B[i];
	if (f<1) B[i]=1-f; else B[i]=0;
}

void errorSquare(float *A,const float *z,const float *x,const float *y,int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	kerrorSquare<<<nbBlock,__CUDA__NBTHREADS>>>(A,z,x,y,N);
	
}


void activationFunctionShrink( const float *A,float *B, int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	functionShrink<<<nbBlock,__CUDA__NBTHREADS>>>(A,B,N);
}


void activationFunctionAdaptedTanH(const float *A,float *B, int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	functionAdaptedTanH<<<nbBlock,__CUDA__NBTHREADS>>>(A,B,N);
}

void activationFunctionLogistic(const float *A,float *B, int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	functionLogistic<<<nbBlock,__CUDA__NBTHREADS>>>(A,B,N);
}

void activationFunctionPShrink(const float *A,float *B, int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	functionPShrink<<<nbBlock,__CUDA__NBTHREADS>>>(A,B,N);
}



void activationFunctionDShrink(const float *A,float *B, int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	functionDShrink<<<nbBlock,__CUDA__NBTHREADS>>>(A,B,N);
}

void activationFunctionDPShrink(const float *A,float *B, int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	functionDPShrink<<<nbBlock,__CUDA__NBTHREADS>>>(A,B,N);
}

void activationFunctionDAdaptedTanH(const float *A,float *B, int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	functionDAdaptedTanH<<<nbBlock,__CUDA__NBTHREADS>>>(A,B,N);
}

void activationFunctionDLogistic(const float *A,float *B, int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	functionDLogistic<<<nbBlock,__CUDA__NBTHREADS>>>(A,B,N);
}


void hingeLossFunction(const float *A,float *B,int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	functionHingeLoss<<<nbBlock,__CUDA__NBTHREADS>>>(A,B,N);
}


__global__ void kmultiplicationTermeATerme(float *A,const float *B,int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) A[i]=A[i]*B[i];
}

void multiplicationTermeATerme(float *A,const float *B,int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	kmultiplicationTermeATerme<<<nbBlock,__CUDA__NBTHREADS>>>(A,B,N);
}

__global__ void k_squareLoss(float *dest,const float *A,const float *B,int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) dest[i]=(A[i]-B[i])*(A[i]-B[i]);
}

__global__ void k_hingeloss(float *dest,const float *A,const float *B,int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) 
	{
		float d=A[i]*B[i];
		if (A[i]*B[i]<1) dest[i]=1-d;
		else dest[i]=0;
	}
}

void GPU_SquareLoss(float *dest,const float *A,const float *B,int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	k_squareLoss<<<nbBlock,__CUDA__NBTHREADS>>>(dest,A,B,N);
}
void GPU_HingeLoss(float *dest,const float *A,const float *B,int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	k_hingeloss<<<nbBlock,__CUDA__NBTHREADS>>>(dest,A,B,N);
}


__global__ void k_fill(float *dest,float v,int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) dest[i]=v;
}

__global__ void k_dhingeloss(float *dest,const float *A,const float *B,int N)
{	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) 
	{
		if (A[i]*B[i]<1) dest[i]=-B[i]; else dest[i]=0;
	}
}



__global__ void k_difference(float *dest,const float *A,const float *B,int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) dest[i]=(A[i]-B[i]);
}


void GPU_Difference(float *dest,const float *A,const float *B,int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	k_difference<<<nbBlock,__CUDA__NBTHREADS>>>(dest,A,B,N);
}

void GPU_DHingeLoss(float *dest,const float *A,const float *B,int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	k_dhingeloss<<<nbBlock,__CUDA__NBTHREADS>>>(dest,A,B,N);
}

__global__ void k_addition(float *dest,const float *A,const float *B,int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) dest[i]=(A[i]+B[i]);
}

void GPU_Addition(float *dest,const float *A,const float *B,int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	k_addition<<<nbBlock,__CUDA__NBTHREADS>>>(dest,A,B,N);
}

__global__ void k_negative(float *dest,const float *A,int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) dest[i]=-A[i];
}


void GPU_Negative(float *dest,const float *A,int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	k_negative<<<nbBlock,__CUDA__NBTHREADS>>>(dest,A,N);
}


void GPU_fill(float *dest,float v,int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	k_fill<<<nbBlock,__CUDA__NBTHREADS>>>(dest,v,N);
}

__global__ void k_square(float *dest,const float *A,int N)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<N) dest[i]=A[i]*A[i];
}

void GPU_Square(float *dest,const float *A,int N)
{
	int nbBlock=N/__CUDA__NBTHREADS+1;
	k_square<<<nbBlock,__CUDA__NBTHREADS>>>(dest,A,N);
}

__global__ void k_l2norm_delta(float *dest,const float *A,const float *B,int nbexamples,int sizeexamples)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<nbexamples*sizeexamples)
	{	
		dest[i]=A[i%nbexamples]*B[i];
	}
}

void GPU_GPUL2Norm_Delta(float *dest,const float *A,const float *B,int nbexamples,int sizeexamples)
{
	int nbBlock=(nbexamples*sizeexamples)/__CUDA__NBTHREADS+1;
	k_l2norm_delta<<<nbBlock,__CUDA__NBTHREADS>>>(dest,A,B,nbexamples,sizeexamples);
}

__global__ void k_sum_parexemple(float *dest,const float *A,int nbexamples,int sizeexamples)
{
	int i = blockIdx.x * __CUDA__NBTHREADS + threadIdx.x;
	if (i<nbexamples*sizeexamples)
	{	
		dest[i%nbexamples]+=A[i];
	}
}

void GPU_Sum_ParExemple(float *out, float *A,int minibatch_size,int size)
{
	int nbBlock=(minibatch_size*size)/__CUDA__NBTHREADS+1;
	k_fill<<<nbBlock,__CUDA__NBTHREADS>>>(out,0.0f,minibatch_size*size);
	k_sum_parexemple<<<nbBlock,__CUDA__NBTHREADS>>>(out,A,minibatch_size,size);

}
