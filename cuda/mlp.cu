
#include <hip/hip_runtime.h>
#include <iostream>
#include "hipblas.h"
//#include "cuPrintf.cu"

//#define __CUDA__NBTHREADS 1024

extern "C"
__global__ void add(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i<n)
    {
        sum[i] = a[i] + b[i];
    }
}

extern "C"
__global__ void GPU_fill(float *dest,float v,int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<N) dest[i]=v;
}


extern "C"
__global__ void setVal(float *dest,float v,int idx)
{
	dest[idx]=v;
}



extern "C"
__global__ void getVal(float *from,float *dest,int idx)
{
	dest[0]=from[idx];
}


extern "C"
__global__ void ccVals(float *from,float **dest,int N)
{
	//# if __CUDA_ARCH__>=200
    //	printf("%d \n", N);
	//#endif 
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i<N) dest[i][0]=from[i];
}

